#include "hip/hip_runtime.h"
#include "common/book.h"

#define N (1024*1024)
#define FULL_DATA_SIZE (N*20)

__global__ void kernel(int* a, int* b, int* c)
{
	int idx = threadIdx.x + blockIdx.x + blockDim.x;
	if (idx < N)
	{
		int idx1 = (idx + 1) % 256;
		int idx2 = (idx + 2) % 256;
		float as = (a[idx] + a[idx1] + a[idx2]) / 3.0f;
		float bs = (b[idx] + b[idx1] + b[idx2]) / 3.0f;
		c[idx] = (as + bs) / 2;
	}
}

int main(void)
{
	hipDeviceProp_t prop;
	int whichDevice;
	hipGetDevice(&whichDevice);
	hipGetDeviceProperties(&prop, whichDevice);

	if (prop.deviceOverlap == 0)
	{
		printf("your device is not support device overlap.\n");
		return 0;
	}

	hipEvent_t start, stop;
	hipStream_t stream;

	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipStreamCreate(&stream);

	int* dev_a, * dev_b, * dev_c;
	int* host_a, * host_b, * host_c;

	hipMalloc((void**)&dev_a, N * sizeof(int));
	hipMalloc((void**)&dev_b, N * sizeof(int));
	hipMalloc((void**)&dev_c, N * sizeof(int));

	hipHostAlloc((void**)&host_a, FULL_DATA_SIZE * sizeof(int), hipHostMallocDefault);
	hipHostAlloc((void**)&host_b, FULL_DATA_SIZE * sizeof(int), hipHostMallocDefault);
	hipHostAlloc((void**)&host_c, FULL_DATA_SIZE * sizeof(int), hipHostMallocDefault);

	for (int i = 0; i < FULL_DATA_SIZE; i++)
	{
		host_a[i] = rand();
		host_b[i] = rand();
	}

	hipEventRecord(start, 0);
	for (int i = 0; i < FULL_DATA_SIZE; i += N)
	{
		hipMemcpyAsync(dev_a, host_a, N * sizeof(N), hipMemcpyHostToDevice, stream);
		hipMemcpyAsync(dev_b, host_b, N * sizeof(N), hipMemcpyHostToDevice, stream);
		kernel << <N / 256, 256, 0, stream >> > (dev_a, dev_b, dev_c);
		hipMemcpyAsync(host_c, dev_c, N * sizeof(N), hipMemcpyDeviceToHost, stream);
	}

	hipStreamSynchronize(stream);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	float elapsedTime;
	hipEventElapsedTime(&elapsedTime, start, stop);
	printf("This Time is: %3.1f ms\n", elapsedTime);

	hipHostFree(host_a);
	hipHostFree(host_b);
	hipHostFree(host_c);
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);
	hipEventDestroy(start);
	hipEventDestroy(stop);
	hipStreamDestroy(stream);
	return 0;
}