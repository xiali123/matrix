#include "hip/hip_runtime.h"
#include "io.h"
#include "utilities.h"
#include <math.h>

#define BLOCK_SIZE 1024

// Parallel SpMV with one Thread per Row
__global__
void parallel_spmv_1(float* values, int* col_idx, int* row_off, float* vect, float* res,
    int m, int n, int nnz) {

    int row = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < m) {
        int begin_index = row_off[row];
        int end_index = row_off[row + 1];

        float row_sum = 0.0;
        for (int i = begin_index; i < end_index; i++) {
            row_sum += (values[i] * vect[col_idx[i]]);
        }

        res[row] = row_sum;
    }

}
////////////////////////////


// Parallel SpMV with one Warp per Row
__global__
void parallel_spmv_2(float* values, int* col_idx, int* row_off, float* vect, float* res,
    int m, int n, int nnz) {

    int thread_id = blockDim.x * blockIdx.x + threadIdx.x;
    int warp_id = thread_id / 32;
    int lane_id = thread_id % 32;

    int row = warp_id;

    if (row < m) {
        int begin_index = row_off[row];
        int end_index = row_off[row + 1];

        float thread_sum = 0.0;
        for (int i = begin_index + lane_id; i < end_index; i += 32)
            thread_sum += values[i] * vect[col_idx[i]];

        thread_sum += __shfl_down(thread_sum, 16);
        thread_sum += __shfl_down(thread_sum, 8);
        thread_sum += __shfl_down(thread_sum, 4);
        thread_sum += __shfl_down(thread_sum, 2);
        thread_sum += __shfl_down(thread_sum, 1);

        if (lane_id == 0)
            res[row] = thread_sum;

    }
}
////////////////////////////

// Parallel SpMV with Average threads per row
__global__
void parallel_spmv_3(float* values, int* col_idx, int* row_off, float* vect, float* res,
    int m, int n, int nnz, int threads_per_row) {

    int thread_id = blockDim.x * blockIdx.x + threadIdx.x;
    int vector_id = thread_id / threads_per_row;
    int lane_id = thread_id % threads_per_row;

    int row = vector_id;

    if (row < m) {
        int begin_index = row_off[row];
        int end_index = row_off[row + 1];

        float thread_sum = 0.0;
        for (int i = begin_index + lane_id; i < end_index; i += threads_per_row)
            thread_sum += values[i] * vect[col_idx[i]];

        int temp = threads_per_row / 2;
        while (temp >= 1) {
            thread_sum += __shfl_down(thread_sum, temp);
            temp /= 2;
        }

        if (lane_id == 0)
            res[row] = thread_sum;

    }
}
////////////////////////////


// Utility function to calculate thread_per_row for parallel_spmv_3 //
int nearest_pow_2(float n) {
    int lg = (int)log2(n);
    return (int)pow(2, lg);
}
////////////////////////////


int main() {

    // Create Cuda Events //
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    ////////////////////////////

    // Reading Dataset //
    int m, n, nnz, nnz_max, nnz_avg, nnz_dev;

    conv(nnz, m, n, nnz_max, nnz_avg, nnz_dev);  // Defined in io.h

    cout << "\nrows    = " << m;
    cout << "\ncolumns = " << n;
    cout << "\nnnz     = " << nnz;
    cout << "\nnnz_max = " << nnz_max;
    cout << "\nnnz_avg = " << nnz_avg;
    cout << "\nnnz_dev = " << nnz_dev;
    cout << "\n\n";

    float* vect = vect_gen(n); //generating dense vector
    ////////////////////////////


    // Serial SpMV //
    float* host_res = new float[m];

    clock_t begin = clock();
    simple_spmv(host_res, vect, values, col_idx, row_off, nnz, m, n);
    clock_t end = clock();
    double cpu_time = double(end - begin) / CLOCKS_PER_SEC;
    cpu_time = cpu_time * 1000;
    ////////////////////////////


    // Device Memory allocation //
    float* d_values, * d_res, * d_vect;
    int* d_row_off, * d_col_idx;
    hipMalloc((void**)&d_values, sizeof(float) * nnz);
    hipMalloc((void**)&d_col_idx, sizeof(int) * nnz);
    hipMalloc((void**)&d_row_off, sizeof(int) * (m + 1));
    hipMalloc((void**)&d_res, sizeof(float) * m);
    hipMalloc((void**)&d_vect, sizeof(float) * n);
    ////////////////////////////


    // Host to device copy //
    hipMemcpy(d_values, values, sizeof(float) * nnz, hipMemcpyHostToDevice);
    hipMemcpy(d_col_idx, col_idx, sizeof(int) * nnz, hipMemcpyHostToDevice);
    hipMemcpy(d_row_off, row_off, sizeof(int) * (m + 1), hipMemcpyHostToDevice);
    hipMemcpy(d_vect, vect, sizeof(float) * n, hipMemcpyHostToDevice);
    ////////////////////////////


    // Parallel SpMV //
    ////////////////////////////
    dim3 dimBlock(BLOCK_SIZE, 1, 1);
    dim3 dimGrid_1((m - 1) / BLOCK_SIZE + 1, 1, 1);
    dim3 dimGrid_2((m - 1) / 32 + 1, 1, 1);
    int threads_per_row = min(32, nearest_pow_2(nnz_avg));
    dim3 dimGrid_3((m - 1) / (1024 / threads_per_row) + 1, 1, 1);

    // Calling one thread per row kernel
    hipEventRecord(start);
    parallel_spmv_1 << <dimGrid_1, dimBlock >> > (d_values, d_col_idx, d_row_off, d_vect, d_res, m, n, nnz);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float gpu_time_1 = 0;
    hipEventElapsedTime(&gpu_time_1, start, stop);

    // calling one warp per row kernel
    hipEventRecord(start);
    parallel_spmv_2 << <dimGrid_2, dimBlock >> > (d_values, d_col_idx, d_row_off, d_vect, d_res, m, n, nnz);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float gpu_time_2 = 0;
    hipEventElapsedTime(&gpu_time_2, start, stop);

    // calling avg threads per row kernel
    hipEventRecord(start);
    parallel_spmv_3 << <dimGrid_3, dimBlock >> > (d_values, d_col_idx, d_row_off, d_vect, d_res, m, n, nnz, threads_per_row);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float gpu_time_3 = 0;
    hipEventElapsedTime(&gpu_time_3, start, stop);

    ////////////////////////////


    // Copy result to host //
    float* result_from_device = new float[m];
    hipMemcpy(result_from_device, d_res, sizeof(float) * n, hipMemcpyDeviceToHost);
    ////////////////////////////

    // Check Result //
    checker(result_from_device, host_res, m);
    ////////////////////////////

    // Free Device Memory //
    hipFree(d_values);
    hipFree(d_col_idx);
    hipFree(d_row_off);
    hipFree(d_res);
    hipFree(d_vect);
    ////////////////////////////

    // Print Statistics //
    cout << "\n\nCPU Execution time                  = " << cpu_time << " ms";
    cout << "\n\nGPU Execution time - Thread per Row = " << gpu_time_1 << " ms";
    cout << "\n\nGPU Execution time - Warp per Row   = " << gpu_time_2 << " ms";
    cout << "\n\n\nThreads per row in avrg per row     = " << threads_per_row;
    cout << "\nGPU Execution time - Avrg per Row   = " << gpu_time_3 << " ms";
    cout << "\n\n";
    ////////////////////////////
}