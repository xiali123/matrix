#include "hip/hip_runtime.h"
#include "common/book.h"

#define imin(a,b) (a>b)?b:a;
#define     N    (33*1024*1024)
const int threadsPerBlock = 256;
const int blocksPerGrid = imin(32,(N+threadsPerBlock-1)/ threadsPerBlock);

struct DataBlock
{
	int deviceIdx;
	float* a;
	float* b;
	int size;
	int offset;
	float returnValue;
};

__global__ void dot(int size, float *a, float *b, float *c)
{
	__shared__ float cache[threadsPerBlock];
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	int stripe = blockDim.x * gridDim.x;
	int cacheIdx = threadIdx.x;
	float temp = 0;
	while (tid < size)
	{
		temp += a[tid] * b[tid];
		tid += stripe;
	}

	cache[cacheIdx] = temp;
	__syncthreads();

	int i = blockDim.x / 2;
	while (i != 0)
	{
		if (cacheIdx < i)
			cache[cacheIdx] += chache[chacheIdx + i];
		__syncthreads();
	}

	if (cacheIdx == 0) c[blockIdx.x] = cache[0];
}

void* runtime_cu(void* pvoidData)
{
	DataBlock* data = (DataBlock*)pvoidData;
	if (data->deviceIdx != 0)
	{
		hipSetDevice(data->deviceIdx);
		hipSetDeviceFlags(hipDeviceMapHost);
	}

	int size = data->size;
	float* a, * b, * c;
	float* dev_a, * dev_b, * dev_c;

	a = data->a;
	b = data->b;
	c = (float*)malloc(sizeof(float) * blocksPerGrid);

	hipHostGetDevicePointer(&dev_a, a, 0);
	hipHostGetDevicePointer(&dev_b, b, 0);
	hipMalloc((void**)&dev_c, blocksPerGrid * sizeof(float));

	dev_a += data->offset;
	dev_b += data->offset;
	dot << <threadsPerBlock, blocksPerGrid >> > (size, dev_a, dev_b, dev_c);

	hipMemcpy(c, dev_c, blocksPerGrid * sizeof(float), hipMemcpyDeviceToHost);

	float returnC = 0;
	for (int i = 0; i < blocksPerGrid; i++)
	{
		returnC += c[i];
	}

	hipFree(dev_c);
	free(c);

	data->returnValue = returnC;
	return 0;
}

int main()
{
	int deviceCount;
	hipGetDeviceCount(&count);
	if (deviceCount < 2)
	{
		printf("this device is just one device.\n");
		return 0;
	}

	hipDeviceProp_t prop;
	for (int i = 0; i < deviceCount; i++)
	{
		hipGetDeviceProperties(&prop, 0);
		if (prop.canMapHostMemory != 1)
		{
			printf("this device is not support MapHostMemoy.\n");
			return 0;
		}
	}
	
	float* a, * b;
	hipSetDevice(0);
	cudaSetDevcieMapFlags(hipDeviceMapHost);
	hipHostAlloc((void**)&a, N * sizeof(float), hipHostMallocWriteCombined | cudaHostAllocProtable | hipHostMallocMapped);
	hipHostAlloc((void**)&b, N * sizeof(float), hipHostMallocWriteCombined | hipHostMallocPortable | hipHostMallocMapped);

	for (int i = 0; i < N; i++)
	{
		a[i] = i;
		b[i] = i * 2;
	}

	DataBlock data[2];
	data[0].deviceIdx = 0;
	data[0].size = N / 2;
	data[0].a = a;
	data[0].b = b;
	data[0].offset = 0;
	
	data[1].deviceIdx = 1;
	data[1].size = N / 2;
	data[1].a = a;
	data[1].b = b;
	data[1].offset = N / 2;
	
	CUTThread thread = start_thread(runtime_cu, &data[1]);
	runtime_cu(&data[0]);
	end_thread(thread);

	hipHostFree(a);
	hipHostFree(b);

	printf("Value calculated:  %f\n", data[0].returnValue + data[1].returnValue);
	return 0;
}