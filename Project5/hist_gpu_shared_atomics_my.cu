#include "hip/hip_runtime.h"
#include "common/book.h"

#define SIZE (40*1024*1024)

__global__ void kernel(int* histo, unsigned char* buffer)
{
	__shared__ int temp[256];
	temp[threadIdx.x] = 0;
	__syncthreads();
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	int striped = blockDim.x * gridDim.x;
	while (tid < SIZE)
	{
		atomicAdd(&(temp[buffer[tid]]), 1);
		tid += striped;
	}
	__syncthreads();
	atomicAdd(&(histo[threadIdx.x]), temp[threadIdx.x]);
}

int main()
{
	unsigned char* buffer = (unsigned char*)big_random_block(SIZE);

	hipDeviceProp_t prop;
	int whichDevice;
	hipGetDevice(&whichDevice);
	hipGetDeviceProperties(&prop, whichDevice);
	int blocks = 3 * prop.multiProcessorCount;

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
	int* histo, * dev_histo;
	unsigned char* dev_buffer;
	histo = (int*)malloc(256 * sizeof(int));
	hipMalloc((void**)&dev_histo, 256 * sizeof(int));
	hipMalloc((void**)&dev_buffer, SIZE * sizeof(unsigned char));

	hipMemcpy(dev_buffer, buffer, SIZE * sizeof(unsigned char), hipMemcpyHostToDevice);
	//hipMemcpy(dev_hsito, hsito, 256 * sizeof(int), hipMemcpyHostToDevice);
	kernel << <blocks, 256 >> > (dev_histo, dev_buffer);
	hipMemcpy(histo, dev_histo, 256 * sizeof(int), hipMemcpyDeviceToHost);

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	float elapsedTime;
	hipEventElapsedTime(&elapsedTime, start, stop);
	printf("average time is : %3.1f ms.\n", elapsedTime);
	for (int i = 0; i < SIZE; i++)
	{
		histo[buffer[i]]--;
	}

	for (int i = 0; i < 256; i++)
	{
		if (histo[i] != 0) printf("Failure \n");
	}
	hipEventDestroy(start);
	hipEventDestroy(stop);
	hipFree(dev_histo);
	hipFree(dev_buffer);
	free(histo);
	free(buffer);
	return 0;
}