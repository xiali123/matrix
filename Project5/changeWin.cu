#include "hip/hip_runtime.h"
#pragma comment (lib, "opengl32.lib")  /* link with Microsoft OpenGL lib */
#pragma comment (lib, "glut64.lib")    /* link with Win64 GLUT lib */
#include <math.h>
#include "common/GL/glut.h"
#include <stdlib.h>

static float angle = 0.0, ratio;
static float x = 0.0f, y = 1.75f, z = 5.0f;
static float lx = 0.0f, ly = 0.0f, lz = -1.0f;
static GLint snowman_display_list;


void changeSize(int w, int h)
{

    // ��ֹ��0��.
    if (h == 0)
        h = 1;

    ratio = 1.0f * w / h;
    // Reset the coordinate system before modifying
    glMatrixMode(GL_PROJECTION);
    glLoadIdentity();

    //�����ӿ�Ϊ�������ڴ�С
    glViewport(0, 0, w, h);

    //���ÿ��ӿռ�
    gluPerspective(45, ratio, 1, 1000);
    glMatrixMode(GL_MODELVIEW);
    glLoadIdentity();
    gluLookAt(x, y, z,
        x + lx, y + ly, z + lz,
        0.0f, 1.0f, 0.0f);
}

void drawSnowMan() {

    glColor3f(1.0f, 1.0f, 1.0f);

    //������
    glTranslatef(0.0f, 0.75f, 0.0f);
    glutSolidSphere(0.75f, 20, 20);


    // ��ͷ
    glTranslatef(0.0f, 1.0f, 0.0f);
    glutSolidSphere(0.25f, 20, 20);

    // ���۾�
    glPushMatrix();
    glColor3f(0.0f, 0.0f, 0.0f);
    glTranslatef(0.05f, 0.10f, 0.18f);
    glutSolidSphere(0.05f, 10, 10);
    glTranslatef(-0.1f, 0.0f, 0.0f);
    glutSolidSphere(0.05f, 10, 10);
    glPopMatrix();

    // ������
    glColor3f(1.0f, 0.5f, 0.5f);
    glRotatef(0.0f, 1.0f, 0.0f, 0.0f);
    glutSolidCone(0.08f, 0.5f, 10, 2);
}

GLuint createDL() {
    GLuint snowManDL;

    //����һ����ʾ�б��
    snowManDL = glGenLists(1);

    // ��ʼ��ʾ�б�
    glNewList(snowManDL, GL_COMPILE);

    // call the function that contains 
    // the rendering commands
    drawSnowMan();

    // endList
    glEndList();

    return(snowManDL);
}

void initScene() {

    glEnable(GL_DEPTH_TEST);
    snowman_display_list = createDL();
}


void renderScene(void) {
    glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);

    //����һ������

    glColor3f(0.9f, 0.9f, 0.9f);
    glBegin(GL_QUADS);
    glVertex3f(-100.0f, 0.0f, -100.0f);
    glVertex3f(-100.0f, 0.0f, 100.0f);
    glVertex3f(100.0f, 0.0f, 100.0f);
    glVertex3f(100.0f, 0.0f, -100.0f);
    glEnd();

    //����36��ѩ��

    for (int i = -3; i < 3; i++)
        for (int j = -3; j < 3; j++) {
            glPushMatrix();
            glTranslatef(i * 10.0, 0, j * 10.0);
            glCallList(snowman_display_list);;
            glPopMatrix();
        }
    glutSwapBuffers();
}
void orientMe(float ang) {

    lx = sin(ang);
    lz = -cos(ang);
    glLoadIdentity();
    gluLookAt(x, y, z,
        x + lx, y + ly, z + lz,
        0.0f, 1.0f, 0.0f);
}

void moveMeFlat(int direction) {
    x = x + direction * (lx) * 0.1;
    z = z + direction * (lz) * 0.1;
    glLoadIdentity();
    gluLookAt(x, y, z,
        x + lx, y + ly, z + lz,
        0.0f, 1.0f, 0.0f);
}


void inputKey(int key, int x, int y) {
    switch (key) {
    case GLUT_KEY_LEFT:
        angle -= 0.01f;
        orientMe(angle); break;
    case GLUT_KEY_RIGHT:
        angle += 0.01f;
        orientMe(angle); break;
    case GLUT_KEY_UP:
        moveMeFlat(1); break;
    case GLUT_KEY_DOWN:
        moveMeFlat(-1); break;
    }
}


int main(int argc, char** argv)
{
    int c = 0;
    char* t = "";
    glutInit(&c, &t);
    glutInitDisplayMode(GLUT_DEPTH | GLUT_DOUBLE | GLUT_RGBA);
    glutInitWindowPosition(100, 100);
    glutInitWindowSize(640, 360);
    glutCreateWindow("SnowMen from 3D-Tech");

    initScene();

    glutSpecialFunc(inputKey);

    glutDisplayFunc(renderScene);
    glutIdleFunc(renderScene);

    glutReshapeFunc(changeSize);

    glutMainLoop();

    return(0);
}