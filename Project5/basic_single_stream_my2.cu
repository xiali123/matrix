#include "hip/hip_runtime.h"
#include "common/book.h"

#define N (1024*1024)
#define FULL_DATA_SIZE (N*20)

__global__ void kernel(int* a, int* b, int* c)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx < N)
	{
		int idx1 = (idx + 1) / 256;
		int idx2 = (idx + 1) / 256;
		float as = (a[idx] + a[idx1] + a[idx2]) / 3.0f;
		float bs = (a[idx] + a[idx1] + a[idx2]) / 3.0f;
		c[idx] = (as + bs) / 2;
	}
}

int main()
{
	hipDeviceProp_t prop;
	int whichDevice;

	hipGetDevice(&whichDevice);
	hipGetDeviceProperties(&prop, whichDevice);
	if (prop.deviceOverlap == 0)
	{
		printf("Device don't support this overlap\n");
		return 0;
	}

	hipEvent_t start, stop;
	hipStream_t stream;

	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipStreamCreate(&stream);
	int* dev_c, * dev_a, * dev_b;
	int* host_a, * host_b, * host_c;

	hipMalloc((void**)&dev_a, N * sizeof(int));
	hipMalloc((void**)&dev_b, N * sizeof(int));
	hipMalloc((void**)&dev_c, N * sizeof(int));

	hipHostAlloc((void**)&host_a, FULL_DATA_SIZE * sizeof(int), hipHostMallocDefault);
	hipHostAlloc((void**)&host_b, FULL_DATA_SIZE * sizeof(int), hipHostMallocDefault);
	hipHostAlloc((void**)&host_c, FULL_DATA_SIZE * sizeof(int), hipHostMallocDefault);

	hipEventRecord(start, 0);
	for (int i = 0; i < FULL_DATA_SIZE; i++)
	{
		host_a[i] = rand();
		host_b[i] = rand();
	}

	for (int i = 0; i < FULL_DATA_SIZE; i += N)
	{
		hipMemcpyAsync(dev_a, host_a, N * sizeof(int), hipMemcpyHostToDevice, stream);
		hipMemcpyAsync(dev_b, host_b, N * sizeof(int), hipMemcpyHostToDevice, stream);
		kernel<<<N/256, 256, 0, stream>>>(dev_a, dev_b, dev_c);
		hipMemcpyAsync(host_c, dev_c, N * sizeof(int), hipMemcpyDeviceToHost, stream);
	}

	hipStreamSynchronize(stream);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	float elapsedTime;
	hipEventElapsedTime(&elapsedTime, start, stop);
	printf("Time is : %3.1f ms\n", elapsedTime);

	hipHostFree(host_a);
	hipHostFree(host_b);
	hipHostFree(host_c);
	hipFree(dev_c);
	hipFree(dev_b);
	hipFree(dev_a);
	hipEventDestroy(start);
	hipEventDestroy(stop);
	hipStreamDestroy(stream);
	return 0;
}