#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "common/book.h"
#include "common/cpu_anim.h"

#define DIM 1024
#define PI 3.1415926535897932f
#define MAX_TEMP 1.0f
#define MIN_TEMP 0.0001f
#define SPEED   0.25f

texture<float> texIn;
texture<float> texOut;
texture<float> texConstStr;

struct DataBlock
{
	unsigned char* output_bitmap;
	CPUAnimBitmap* bitmap;
	float* dev_outStr;
	float* dev_inStr;
	float* dev_constStr;
	hipEvent_t start, stop;
	float totalTime;
	float frames;
};


__global__ void copy_const_kernel(float* iptr) 
{
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	int offset = x + y * blockDim.x * gridDim.x;

	float c = tex1Dfetch(texConstStr, offset);
	if (c != 0)
	{
		iptr[offset] = c;
	}
}

__global__ void blend_kernel(float* dst,
	bool dstOut) {
	// map from threadIdx/BlockIdx to pixel position
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	int offset = x + y * blockDim.x * gridDim.x;

	int left = offset - 1;
	int right = offset + 1;
	if (x == 0)   left++;
	if (x == DIM - 1) right--;

	int top = offset - DIM;
	int bottom = offset + DIM;
	if (y == 0)   top += DIM;
	if (y == DIM - 1) bottom -= DIM;

	float   t, l, c, r, b;
	if (dstOut) {
		t = tex1Dfetch(texIn, top);
		l = tex1Dfetch(texIn, left);
		c = tex1Dfetch(texIn, offset);
		r = tex1Dfetch(texIn, right);
		b = tex1Dfetch(texIn, bottom);

	}
	else {
		t = tex1Dfetch(texOut, top);
		l = tex1Dfetch(texOut, left);
		c = tex1Dfetch(texOut, offset);
		r = tex1Dfetch(texOut, right);
		b = tex1Dfetch(texOut, bottom);
	}
	dst[offset] = c + SPEED * (t + b + r + l - 4 * c);
}

void anim_gpu(DataBlock* d, int ticks) {
	HANDLE_ERROR(hipEventRecord(d->start, 0));
	dim3    blocks(DIM / 16, DIM / 16);
	dim3    threads(16, 16);
	CPUAnimBitmap* bitmap = d->bitmap;

	// since tex is global and bound, we have to use a flag to
	// select which is in/out per iteration
	volatile bool dstOut = true;
	for (int i = 0; i < 90; i++) {
		float* in, * out;
		if (dstOut) {
			in = d->dev_inStr;
			out = d->dev_outStr;
		}
		else {
			out = d->dev_inStr;
			in = d->dev_outStr;
		}
		copy_const_kernel << <blocks, threads >> > (in);
		blend_kernel << <blocks, threads >> > (out, dstOut);
		dstOut = !dstOut;
	}
	float_to_color << <blocks, threads >> > (d->output_bitmap,
		d->dev_inStr);

	HANDLE_ERROR(hipMemcpy(bitmap->get_ptr(),
		d->output_bitmap,
		bitmap->image_size(),
		hipMemcpyDeviceToHost));

	HANDLE_ERROR(hipEventRecord(d->stop, 0));
	HANDLE_ERROR(hipEventSynchronize(d->stop));
	float   elapsedTime;
	HANDLE_ERROR(hipEventElapsedTime(&elapsedTime,
		d->start, d->stop));
	d->totalTime += elapsedTime;
	++d->frames;
	printf("Average Time per frame:  %3.1f ms\n",
		d->totalTime / d->frames);
}


void anim_exit(DataBlock* d)
{
	hipUnbindTexture(texIn);
	hipUnbindTexture(texOut);
	hipUnbindTexture(texConstStr);

	hipFree(d->dev_inStr);
	hipFree(d->dev_outStr);
	hipFree(d->dev_constStr);

	hipEventDestroy(d->start);
	hipEventDestroy(d->stop);
}


int main()
{
	DataBlock data;
	CPUAnimBitmap bitmap(DIM, DIM, &data);
	data.bitmap = &bitmap;
	data.totalTime = 0;
	data.frames = 0;

	hipEventCreate(&data.start);
	hipEventCreate(&data.stop);

	int imageSize = bitmap.image_size();

	hipMalloc((void**)&data.output_bitmap, imageSize);
	hipMalloc((void**)&data.dev_inStr, imageSize);
	hipMalloc((void**)&data.dev_outStr, imageSize);
	hipMalloc((void**)&data.dev_constStr, imageSize);
	
	hipBindTexture(NULL, texConstStr, data.dev_constStr, imageSize);
	hipBindTexture(NULL, texIn, data.dev_inStr, imageSize);
	hipBindTexture(NULL, texOut, data.dev_outStr, imageSize);

	float* temp = (float*)malloc(imageSize);
	for (int i = 0; i < DIM * DIM; i++) {
		temp[i] = 0;
		int x = i % DIM;
		int y = i / DIM;
		if ((x > 300) && (x < 600) && (y > 310) && (y < 601))
			temp[i] = MAX_TEMP;
	}
	temp[DIM * 300 + 200] = (MAX_TEMP + MIN_TEMP) / 2;
	temp[DIM * 600 + 700] = MIN_TEMP;
	temp[DIM * 300 + 300] = MIN_TEMP;
	temp[DIM * 600 + 300] = MIN_TEMP;
	for (int y = 800; y < 900; y++) {
		for (int x = 400; x < 500; x++) {
			temp[x + y * DIM] = MIN_TEMP;
		}
	}

	hipMemcpy(data.dev_constStr, temp, imageSize, hipMemcpyHostToDevice);

	for (int y = 800; y < DIM; y++) {
		for (int x = 0; x < 200; x++) {
			temp[x + y * DIM] = MAX_TEMP;
		}
	}

	hipMemcpy(data.dev_inStr, temp, imageSize, hipMemcpyHostToDevice);
	free(temp);

	bitmap.anim_and_exit((void (*)(void*, int))anim_gpu,
		(void (*)(void*))anim_exit);
}