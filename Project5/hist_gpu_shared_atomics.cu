#include "hip/hip_runtime.h"
#include "common/book.h"

#define SIZE    (100*1024*1024)
__global__ void kernel(int* histo, unsigned char* buffer)
{
	__shared__ int temp[256];
	temp[threadIdx.x] = 0;
	__syncthreads();
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	int striped = blockDim.x * gridDim.x;
	while (tid < SIZE)
	{
		atomicAdd(&(temp[buffer[tid]]), 1);
		tid += striped;
	}

	__syncthreads();
	atomicAdd(&(histo[threadIdx.x]), temp[threadIdx.x]);
}

int main()
{
	unsigned char* buffer = (unsigned char*)big_random_block(SIZE);
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	int* histo, *dev_histo;
	unsigned char* dev_buffer;
	histo = (int*)malloc(256 * sizeof(int));
	hipMalloc((void**)&dev_histo, 256 * sizeof(int));
	hipMalloc((void**)&dev_buffer, SIZE * sizeof(char));
	hipMemcpy(dev_buffer, buffer, SIZE, hipMemcpyHostToDevice);

	hipDeviceProp_t prop;
	int dev;
	hipGetDevice(&dev);
	hipGetDeviceProperties(&prop, dev);
	int blocks = 2 * prop.multiProcessorCount;
	kernel << <blocks, 256 >> > (dev_histo, dev_buffer);

	hipMemcpy(histo, dev_histo, 256 * sizeof(int), hipMemcpyDeviceToHost);

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	float elapsedTime;
	hipEventElapsedTime(&elapsedTime, start, stop);
	printf("average time is : %3.1f ms \n", elapsedTime);

	for (int i = 0; i < SIZE; i++)
	{
		histo[buffer[i]]--;
	}

	for (int i = 0; i < 256; i++)
	{
		if (histo[i] != 0)
		{
			printf("this is not true.\n");
		}
	}

	hipFree(dev_histo);
	hipFree(dev_buffer);
	hipEventDestroy(start);
	hipEventDestroy(stop);
	free(buffer);
	free(histo);
	return 0;
}