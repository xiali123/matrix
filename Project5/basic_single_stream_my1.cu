#include "hip/hip_runtime.h"
#include "common/book.h"

#define N (1024*1024)
#define FULL_DATA_SIZE (N*20)

__global__ void kernel(int* dev_a, int* dev_b, int* dev_c)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if(idx < N)
	{
		int idx1 = (idx + 1) % 256;
		int idx2 = (idx + 2) % 256;
		float as = (dev_a[idx] + dev_a[idx1] + dev_a[idx2]) / 3.0f;
		float bs = (dev_b[idx] + dev_b[idx1] + dev_b[idx2]) / 3.0f;
		dev_c[idx] = (as + bs) / 2;
	}
}

int main(void)
{
	hipDeviceProp_t prop;
	int whichDevice;

	hipGetDevice(&whichDevice);
	hipGetDeviceProperties(&prop, whichDevice);

	if (!prop.deviceOverlap)
	{
		printf("Device will not handle overlaps, so no speed up from streams\n");
		return 0;
	}

	hipEvent_t start, stop;
	hipStream_t stream;

	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipStreamCreate(&stream);
	
	int* dev_a, * dev_b, * dev_c;
	int* host_a, * host_b, * host_c;

	hipMalloc((void**)&dev_a, N * sizeof(int));
	hipMalloc((void**)&dev_b, N * sizeof(int));
	hipMalloc((void**)&dev_c, N * sizeof(int));

	hipHostAlloc((void**)&host_a, FULL_DATA_SIZE * sizeof(int), hipHostMallocDefault);
	hipHostAlloc((void**)&host_b, FULL_DATA_SIZE * sizeof(int), hipHostMallocDefault);
	hipHostAlloc((void**)&host_c, FULL_DATA_SIZE * sizeof(int), hipHostMallocDefault);

	for (int i = 0; i < FULL_DATA_SIZE; i++)
	{
		host_a[i] = rand();
		host_b[i] = rand();
	}

	hipEventRecord(start, 0);

	for (int i = 0; i < FULL_DATA_SIZE; i += N)
	{
		hipMemcpyAsync(dev_a, host_a + i, N * sizeof(int), hipMemcpyHostToDevice, stream);
		hipMemcpyAsync(dev_b, host_b + i, N * sizeof(int), hipMemcpyHostToDevice, stream);

		kernel << <N / 256, 256, 0, stream >> > (dev_a, dev_b, dev_c);
		hipMemcpyAsync(host_c + i, dev_c, N * sizeof(int), hipMemcpyDeviceToHost, stream);
	}


	hipStreamSynchronize(stream);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	
	float elapsedTime;

	hipEventElapsedTime(&elapsedTime, start, stop);

	printf("Time taken:  %3.1f ms\n", elapsedTime);

	hipHostFree(host_a);
	hipHostFree(host_b);
	hipHostFree(host_c);
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);

	hipEventDestroy(start);
	hipEventDestroy(stop);
	hipStreamDestroy(stream);
	return 0;
}