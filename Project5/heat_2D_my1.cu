#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "common/book.h"
#include "common/cpu_anim.h"

#define DIM 1024
#define MIN_TEMP 0.00001f
#define MAX_TEMP 1.0f
#define SPEED 0.25f

texture<float, 2> texIn;
texture<float, 2> texOut;
texture<float, 2> texConstStr;

struct DataBlock
{
	unsigned char* output_bitmap;
	float* dev_inStr;
	float* dev_outStr;
	float* dev_constStr;

	CPUAnimBitmap* bitmap;
	hipEvent_t start, stop;
	float totalTime;
	float frames;
};

__global__ void copy_const_kernel(float* iptr)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	int offset = x + y * blockDim.x * gridDim.x;

	float c = tex2D(texConstStr, x, y);
	if (c != 0) iptr[offset] = c;
}

__global__ void blend_kernel(float* dst, bool dstOut)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	int offset = x + y * blockDim.x * gridDim.x;

	float l, r, t, b, c;
	if (dstOut)
	{
		l = tex2D(texIn, x - 1, y);
		r = tex2D(texIn, x + 1, y);
		t = tex2D(texIn, x, y - 1);
		b = tex2D(texIn, x, y + 1);
		c = tex2D(texIn, x, y);
	}
	else
	{
		l = tex2D(texOut, x - 1, y);
		r = tex2D(texOut, x + 1, y);
		t = tex2D(texOut, x, y - 1);
		b = tex2D(texOut, x, y + 1);
		c = tex2D(texOut, x, y);
	}
	dst[offset] = c + SPEED * (l + r + t + b - 4 * c);
}

void anim_gpu(DataBlock* d, int intr)
{
	hipEventRecord(d->start, 0);
	dim3 blocks(DIM / 16, DIM / 16);
	dim3 threads(16, 16);
	CPUAnimBitmap* bitmap = d->bitmap;
	volatile bool dstOut = true;
	for (int i = 0; i < 90; i++)
	{
		float* in, * out;
		if (dstOut)
		{
			in = d->dev_inStr;
			out = d->dev_outStr;
		}
		else
		{
			in = d->dev_outStr;
			out = d->dev_inStr;
		}
		copy_const_kernel << <blocks, threads >> > (in);
		blend_kernel << <blocks, threads >> > (out, dstOut);
		dstOut = !dstOut;
	}

	float_to_color << <blocks, threads >> > (d->output_bitmap, d->dev_inStr);
	hipMemcpy(bitmap->get_ptr(), d->output_bitmap, bitmap->image_size(), hipMemcpyDeviceToHost);

	hipEventRecord(d->stop, 0);
	hipEventSynchronize(d->stop);
	float elapsedTime;
	hipEventElapsedTime(&elapsedTime, d->start, d->stop);
	d->totalTime += elapsedTime;
	++d->frames;
	printf("Average time is : %3.1f ms\n", d->totalTime / d->frames);
}

void anim_exit(DataBlock* d)
{
	hipUnbindTexture(texIn);
	hipUnbindTexture(texOut);
	hipUnbindTexture(texConstStr);

	hipFree(d->dev_inStr);
	hipFree(d->dev_outStr);
	hipFree(d->dev_constStr);

	hipEventDestroy(d->start);
	hipEventDestroy(d->stop);
}

int main()
{
	DataBlock data;
	CPUAnimBitmap bitmap(DIM, DIM, &data);
	data.bitmap = &bitmap;
	data.totalTime = 0;
	data.frames = 0;

	hipEventCreate(&data.start);
	hipEventCreate(&data.stop);

	int imageSize = bitmap.image_size();

	hipMalloc((void**)&data.output_bitmap, imageSize);
	hipMalloc((void**)&data.dev_inStr, imageSize);
	hipMalloc((void**)&data.dev_outStr, imageSize);
	hipMalloc((void**)&data.dev_constStr, imageSize);

	hipChannelFormatDesc desc = hipCreateChannelDesc<float>();
	HANDLE_ERROR(hipBindTexture2D(NULL, texConstStr,
		data.dev_constStr,
		desc, DIM, DIM,
		sizeof(float) * DIM));

	HANDLE_ERROR(hipBindTexture2D(NULL, texIn,
		data.dev_inStr,
		desc, DIM, DIM,
		sizeof(float) * DIM));

	HANDLE_ERROR(hipBindTexture2D(NULL, texOut,
		data.dev_outStr,
		desc, DIM, DIM,
		sizeof(float) * DIM));

	float* temp = (float*)malloc(imageSize);
	for (int i = 0; i < DIM * DIM; i++)
	{
		temp[i] = 0; 
		int x = i % DIM;
		int y = i / DIM;
		if (x > 300 && x < 600 && y > 300 && y < 600) temp[i] = MAX_TEMP;
	}

	temp[DIM * 100 + 100] = (MAX_TEMP + MIN_TEMP) / 2;
	temp[DIM * 700 + 100] = MIN_TEMP;
	temp[DIM * 300 + 300] = MIN_TEMP;
	temp[DIM * 200 + 700] = MIN_TEMP;

	for (int y = 800; y < 900; y++)
	{
		for (int x = 400; x < 500; x++)
		{
			temp[x + y * DIM] = MIN_TEMP;
		}
	}

	hipMemcpy(data.dev_constStr, temp, imageSize, hipMemcpyHostToDevice);
	for (int y = 800; y < DIM; y++)
	{
		for (int x = 0; x < 200; x++) temp[x + y * DIM] = MAX_TEMP;
	}
	hipMemcpy(data.dev_inStr, temp, imageSize, hipMemcpyHostToDevice);
	free(temp);
	bitmap.anim_and_exit((void(*)(void*, int))anim_gpu, (void(*)(void*))anim_exit);
	return 0;
}